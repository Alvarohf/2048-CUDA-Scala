#include "hip/hip_runtime.h"
﻿/*====================================================================================
|                 Luis Alejandro Cabanillas Prudencio                                |
|                     Álvaro de las Heras Fernández                                  |
|                                                                                    |
|         16384 Juego que simula al 2048 implementado con matrices y CUDA            |
====================================================================================*/

//Librerias de CUDA
#include "hip/hip_runtime.h" 
#include "" 
#include "hiprand/hiprand_kernel.h"
#include <hip/hip_runtime.h>

//Librerias de C y C++
#include <stdio.h>
#include <time.h>
#include <conio.h>
#include <stdlib.h>
#include <string.h>
#include <windows.h>
#define TILE_WIDTH 2

//Cabeceras de las funciones implmentadas en CPU
int generarNumeros(bool dificultad);
int* inicializarMatriz(int p_num_columnas, int p_num_filas, bool dificultad, int p_semillas_alta, int p_semillas_baja);
void imprimirMatrizVector(int* p_matriz_vector, int p_num_columnas, int p_num_filas);
bool contiene(int p_valor, int* semillas, int p_tam);
int* generaIndexSemilla(int p_num_columnas, int p_num_filas, int p_tam);
void modoManual(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_baja, int p_semillas_alta);
void modoAutomatico(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_baja, int p_semillas_alta);
void guardarPartida(int* tablero, int vidas, int puntuacion, int p_num_columnas, int p_num_filas);
int* cargarPartida(int* p_vidas, int* p_puntuacion, int* p_num_columnas, int* p_num_filas);
int comprobarCasillasVacias(int * p_tablero, int p_num_columnas, int p_num_filas);
int* rellenarTablero(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_alta, int p_semillas_baja, bool* p_lleno);
void obtenerCaracteristicasCUDA(int p_num_columnas, int p_num_filas);
void Color(int fondo, int fuente);
void guardarRecord(int p_record);
int cargarRecord();

//Funciones que se ejecutaran en la GPU de NVIDIA
/**
* Funcion que mueve los valores y los suma (si son iguales) hacia abajo moviendo los 0 a la zona superior
* @param M_dev Tablero con los datos sobre el que se realizaran las operaciones
* @param p_num_columnas Numero de columnas del tablero
* @param p_num_filas Numero de filas del tablero
*/
__global__ void  moverAbajo(int* M_dev, int p_num_columnas, int p_num_filas, int* p_puntuacion_dev) {
	//Obtendra la suma que se añadir a la puntuacion
	int suma = 0;
	//Primero quitamos los 0 de la matriz para ello cada hilo ira bajando los valores
	//de la matriz de arriba si no son 0 esto se hara varias veces hasta asegurarse de que este completo
	for (int i = 0; i < p_num_filas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if (M_dev[threadIdx.x] == 0) {
			//Hay que asegurarse de no pasarse de rango de la columna
			if (threadIdx.x >= p_num_columnas)
				//Si es distinto de 0 el valor anterior se procede a bajar
				if (!(M_dev[threadIdx.x - p_num_columnas] == 0)) {
					//Se reemplaza el valor de la casilla actual por el anterior
					M_dev[threadIdx.x] = M_dev[threadIdx.x - p_num_columnas];
					//Se deja la casilla anterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x - p_num_columnas] = 0;
				}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}
	//Una vez tenemos juntos los valores se suman los valores contiguos si son iguales
		//Al igual que antes comenzamos por el ultimo valor
	for (int k = (p_num_filas - 1); k > 0; k--) {
		//Nos aseguramos de que haya un hilo por columna
		if ((threadIdx.x / p_num_columnas) == k) {
			//Ademas vemos que no se accedan a zonas que puedan dar fallo
			if (threadIdx.x >= p_num_columnas) {
				//Se comprueba que la casilla no sea 0
				if (M_dev[threadIdx.x - p_num_columnas] >= 0) {
					//Se comprueba si son iguales para sumar valores
					if (M_dev[threadIdx.x] == M_dev[threadIdx.x - p_num_columnas])
					{
						suma = M_dev[threadIdx.x] * 2;
						//Si son iguales se suman los valores dando como resultado el doble
						M_dev[threadIdx.x] = M_dev[threadIdx.x] * 2;
						//La casilla anterior se deja a 0
						M_dev[threadIdx.x - p_num_columnas] = 0;
					}
				}
			}
		}
		//Se sincronizan los hilos en cada suma
		__syncthreads();
	}
	//Se vuelven a quitar los 0 tras sumar para dejar un resultado mas agradable
	//Primero quitamos los 0 de la matriz para ello cada hilo ira bajando los valores
	//de la matriz de arriba si no son 0 esto se hara varias veces hasta asegurarse de que este completo
	for (int i = 0; i < p_num_filas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Hay que asegurarse de no pasarse de rango de la columna
			if (threadIdx.x >= (p_num_columnas))
				//Si es distinto de 0 el valor anterior se procede a bajar
				if (!(M_dev[threadIdx.x - p_num_columnas] == 0)) {
					//Se reemplaza el valor de la casilla actual por el anterior
					M_dev[threadIdx.x] = M_dev[threadIdx.x - (p_num_columnas)];
					//Se deja la casilla anterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x - (p_num_columnas)] = 0;
				}
		}
	}
	//Para obtener la puntuacion nos aseguramos de cada hilo sume en orden la suma de puntos a la variable
	for (int i = 0; i < (p_num_columnas*p_num_filas); i++) {
		if (threadIdx.x == i) {
			*p_puntuacion_dev = *p_puntuacion_dev + suma;
		}
		//Hacemos que esperen a la ejecucion de la instruccion
		__syncthreads();
	}
}
/**
* Funcion que mueve los valores y los suma (si son iguales) hacia arriba moviendo los 0 a la zona inferior
* @param M_dev Tablero con los datos sobre el que se realizaran las operaciones
* @param p_num_columnas Numero de columnas del tablero
* @param p_num_filas Numero de filas del tablero
*/
__global__ void  moverArriba(int* M_dev, int p_num_columnas, int p_num_filas, int* p_puntuacion_dev) {
	//Obtendra la suma que se añadir a la puntuacion
	int suma = 0;
	//Primero quitamos los 0 de la matriz para ello cada hilo ira subiendo los valores
	//de la matriz de abajo si no son 0 esto se hara varias veces hasta asegurarse de que este completo
	for (int i = 0; i < p_num_filas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Hay que asegurarse de no pasarse de rango de la columna
			if ((threadIdx.x + p_num_columnas) <= (p_num_columnas*p_num_filas))
				//Si es distinto de 0 el valor anterior se procede a bajar
				if (!(M_dev[threadIdx.x + p_num_columnas] == 0)) {

					//Se reemplaza el valor de la casilla actual por el de despues
					M_dev[threadIdx.x] = M_dev[threadIdx.x + (p_num_columnas)];
					//Se deja la casilla posterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x + (p_num_columnas)] = 0;
				}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}

	//Sumamos los numeros
	//Para ello tenemos que tener cuidado con la concurrencia por lo que se deja 1 hilo por
	//columna ademas se empieza de menor hilo a mayor hilo
	for (int k = 0; k < p_num_filas; k++) {
		//Nos aseguramos de que sean hilos de distintas columnas
		if ((threadIdx.x / p_num_columnas) == k) {
			//Nos aseguramos de que se encuentre en memoria
			if (M_dev[threadIdx.x + p_num_columnas] >= 0) {
				//Si son iguales se procede a la suma
				if (M_dev[threadIdx.x] == M_dev[threadIdx.x + p_num_columnas])
				{
					suma = M_dev[threadIdx.x] * 2;
					//La casilla actual tendra el doble del valor
					M_dev[threadIdx.x] = M_dev[threadIdx.x] * 2;
					//La casilla posterior se pone a 0
					M_dev[threadIdx.x + p_num_columnas] = 0;
				}
			}
		}
		__syncthreads();
	}
	//Se repite el proceso para dejar el resultado sin 0 intermedios
	//Primero quitamos los 0 de la matriz para ello cada hilo ira subiendo los valores
	//de la matriz de abajo si no son 0 esto se hara varias veces hasta asegurarse de que este completo
	for (int i = 0; i < p_num_filas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Hay que asegurarse de no pasarse de rango de la columna
			if (threadIdx.x >= (p_num_columnas))
				//Si es distinto de 0 el valor anterior se procede a bajar
				if (!(M_dev[threadIdx.x + p_num_columnas] == 0)) {
					//Se reemplaza el valor de la casilla actual por el de despues
					M_dev[threadIdx.x] = M_dev[threadIdx.x + (p_num_columnas)];
					//Se deja la casilla posterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x + (p_num_columnas)] = 0;
				}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}
	//Para obtener la puntuacion nos aseguramos de cada hilo sume en orden la suma de puntos a la variable
	for (int i = 0; i < (p_num_columnas*p_num_filas); i++) {
		if (threadIdx.x == i) {
			*p_puntuacion_dev = *p_puntuacion_dev + suma;
		}
		//Hacemos que esperen a la ejecucion de la instruccion
		__syncthreads();
	}
}
__global__ void  moverIzquierda(int* M_dev, int p_num_columnas, int p_num_filas, int* p_puntuacion_dev) {
	//Obtendra la suma que se añadir a la puntuacion
	int suma = 0;
	//Movemos los valores distintos de 0 a la izquierda
	for (int i = 0; i < p_num_columnas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Si es distinto de 0 el valor anterior se procede a bajar siempre y cuando sea de la misma fila
			if (!(M_dev[threadIdx.x + 1] <= 0) && ((threadIdx.x + 1) < p_num_columnas*((threadIdx.x / p_num_columnas) + 1))) {
				//Se reemplaza el valor de la casilla actual por el de despues
				M_dev[threadIdx.x] = M_dev[threadIdx.x + 1];
				//Se deja la casilla posterior a 0 para que pueda ser ocupada por otros valores
				M_dev[threadIdx.x + 1] = 0;
			}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}

	//Sumamos los numeros dejando 1 acceso al hilo por fila
	for (int k = 0; k < p_num_columnas; k++) {
		if ((threadIdx.x % p_num_columnas) == k) {
			//Si el valor es mayor o distinto de 0 se hace
			if (M_dev[threadIdx.x + 1] >= 0) {
				//Si los valores son iguales se suma
				if ((threadIdx.x + 1) < p_num_columnas*((threadIdx.x / p_num_columnas) + 1)) {


					if (M_dev[threadIdx.x] == M_dev[threadIdx.x + 1])
					{
						suma = M_dev[threadIdx.x] * 2;
						//Se suma a la casilla actual
						M_dev[threadIdx.x] = M_dev[threadIdx.x] * 2;
						//Se deja la posterior a 0
						M_dev[threadIdx.x + 1] = 0;
					}
				}
			}
		}
		//Sincronizamos los hilos
		__syncthreads();
	}
	//Volvemos a mover los 0 para dejar un resultado mas agradable
	//Movemos los valores distintos de 0 a la izquierda
	for (int i = 0; i < p_num_columnas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Si es distinto de 0 el valor anterior se procede a bajar siempre y cuando sea de la misma fila
			if (!(M_dev[threadIdx.x + 1] <= 0) && ((threadIdx.x + 1) < p_num_columnas*((threadIdx.x / p_num_columnas) + 1))) {
				//Se reemplaza el valor de la casilla actual por el de despues
				M_dev[threadIdx.x] = M_dev[threadIdx.x + 1];
				//Se deja la casilla posterior a 0 para que pueda ser ocupada por otros valores
				M_dev[threadIdx.x + 1] = 0;
			}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}
	//Para obtener la puntuacion nos aseguramos de cada hilo sume en orden la suma de puntos a la variable
	for (int i = 0; i < (p_num_columnas*p_num_filas); i++) {
		if (threadIdx.x == i) {
			*p_puntuacion_dev = *p_puntuacion_dev + suma;
		}
		//Hacemos que esperen a la ejecucion de la instruccion
		__syncthreads();
	}
}
__global__ void  moverDerecha(int* M_dev, int p_num_columnas, int p_num_filas, int* p_puntuacion_dev) {
	//Obtendra la suma que se añadir a la puntuacion
	int suma = 0;
	//Movemos los valores distintos de 0 a la derecha
	for (int i = 0; i < p_num_columnas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Si es distinto de 0 el valor anterior se procede a bajar siempre y cuando sea de la misma fila
			if (threadIdx.x >= 1) {
				if (!(M_dev[threadIdx.x - 1] <= 0) && ((threadIdx.x - 1) >= p_num_columnas * ((threadIdx.x / p_num_columnas)))) {
					//Se reemplaza el valor de la casilla actual por el de despues
					M_dev[threadIdx.x] = M_dev[threadIdx.x - 1];
					//Se deja la casilla posterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x - 1] = 0;
				}
			}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}
	//Sumamos los numeros hacia la derecha
	//Para ello dejamos que cada hilo acceda a una unica fila
	for (int k = (p_num_columnas); k > 0; k--) {
		//Eso se consigue haciendo el resto
		if ((threadIdx.x % p_num_columnas) == k) {

			//Nos aseguramos de que este dentro de la matriz
			if (M_dev[threadIdx.x - 1] >= 0) {
				//Si son iguales procedemos a sumar
				if (M_dev[threadIdx.x] == M_dev[threadIdx.x - 1])
				{
					suma = M_dev[threadIdx.x] * 2;
					M_dev[threadIdx.x] = M_dev[threadIdx.x] * 2;
					M_dev[threadIdx.x - 1] = 0;
				}
			}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos antes de empezar los siguientes
		__syncthreads();
	}	for (int i = 0; i < p_num_columnas; i++) {
		//En caso de que el valor del hilo es 0 se mueve
		if ((M_dev[threadIdx.x] == 0)) {
			//Si es distinto de 0 el valor anterior se procede a desplazar siempre y cuando sea de la misma fila
			if (threadIdx.x >= 1) {
				if (!(M_dev[threadIdx.x - 1] <= 0) && ((threadIdx.x - 1) >= p_num_columnas * ((threadIdx.x / p_num_columnas)))) {
					//Se reemplaza el valor de la casilla actual por el de antes
					M_dev[threadIdx.x] = M_dev[threadIdx.x - 1];
					//Se deja la casilla anterior a 0 para que pueda ser ocupada por otros valores
					M_dev[threadIdx.x - 1] = 0;
				}
			}
		}
		//Sincronizamos los hilos para  asegurarnos de que acaban juntos
		__syncthreads();
	}
	//Para obtener la puntuacion nos aseguramos de cada hilo sume en orden la suma de puntos a la variable
	for (int i = 0; i < (p_num_columnas*p_num_filas); i++) {
		if (threadIdx.x == i) {
			*p_puntuacion_dev = *p_puntuacion_dev + suma;
		}
		//Hacemos que esperen a la ejecucion de la instruccion
		__syncthreads();
	}
}
//Codigo main que lanzara el programa
int main(int argc, char** argv)
{
	//Ancho de la matriz (EJE X)
	int num_columnas = 3;
	//Alto de la matriz (EJE Y)
	int num_filas = 3;
	//Numero de semillas segun dificultad
	int SEMILLAS_ALTA = 8;
	int SEMILLAS_BAJA = 15;
	int* tablero;
	//Modo de dificultad
	bool dificultad = false;
	printf("¡Bienvenido al juego 16384!\n");
	//Si hay suficientes argumentos coge los valores pasados por consola
	if (argc > 3) {
		//Hacemos un cast a entero del valor de la consola
		dificultad = (2 == atoi(argv[2]));
		num_columnas = atoi(argv[3]);
		num_filas = atoi(argv[4]);
		tablero = inicializarMatriz(num_columnas, num_filas, dificultad, SEMILLAS_ALTA, SEMILLAS_BAJA);
		obtenerCaracteristicasCUDA(num_columnas, num_filas);
		getch();
		//Modo automatico y modo manual
		if (strcmp(argv[1], "-a") == 0) {
			modoAutomatico(tablero, num_columnas, num_filas, dificultad, SEMILLAS_ALTA, SEMILLAS_BAJA);
		}
		else if (strcmp(argv[1], "-m") == 0)
		{
			modoManual(tablero, num_columnas, num_filas, dificultad, SEMILLAS_ALTA, SEMILLAS_BAJA);
		}
		else {
			printf("El parametro que ha introducido %s no es valido\n", argv[1]);
		}
	}
	
	return 0;
}
/**
* Imprime un vector (que representa una matriz) como si fuera una matriz
*@param p_matriz_vector Vector que se mostrara por pantalla
*@param P_WIDTH_X Anchura de la matriz
*@param P_WIDTH_Y Altura de la matriz
*/
void imprimirMatrizVector(int* p_matriz_vector, int p_num_columnas, int p_num_filas) {
	//Bucle para imprimir las filas y colummnas
	for (int i = 0; i < p_num_filas; i++) {
		for (int j = 0; j < p_num_columnas; j++) {

			switch (p_matriz_vector[i * p_num_columnas + j]) {//	Modifica el color en el que se mostrarán los elementos
			case 0:
				Color(8, 8);
				break;
			case 2:
				Color(15, 0);
				break;
			case 4:
				Color(14, 0);
				break;
			case 8:
				Color(13, 0);
				break;
			case 16:
				Color(5, 15);
				break;
			case 32:
				Color(6, 0);
				break;
			case 64:
				Color(4, 0);
				break;
			case 128:
				Color(9, 0);
				break;
			case 256:
				Color(1, 0);
				break;
			case 512:
				Color(10, 0);
				break;
			case 1024:
				Color(2, 0);
				break;
			case 2048:
				Color(7, 0);
				break;
			case 4096:
				Color(8, 0);
				break;
			case 8192:
				Color(3, 0);
				break;
			case 16384:
				Color(15, 0);
				break;
			default:
				Color(0, 15);
				break;
			}
			printf("%d", p_matriz_vector[i * p_num_columnas + j]);
			printf("\t");
			Color(0,15);

		}

		printf("\n");

	}
	Color(0, 15);
}
/**
* Inicializa la matriz con multiplos de dos en funcion de la dificultad y semillas
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
* @param p_dificultad indica el modo de dificultad elegido
* @param p_semillas_alta numero de semillas que se crearan en nivel de juego alto
* @param p_semillas_baja numero de semillas que se crearan en nivel de juego bajo
*/
int* inicializarMatriz(int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_alta, int p_semillas_baja) {
	//Matriz que representa el tablero que se inicializa ademas se le asigna memoria
	int* tablero = (int *)malloc(p_num_columnas*p_num_filas * sizeof(int));
	//Numero de semillas que se generaran
	int numero_semillas = p_semillas_baja;
	//Vector con las posiciones de las semillas que contendran un valor inicial
	int* posicion_semillas;
	//Instante de tiempo que se usara para obtener numeros aleatorios
	srand((unsigned int)time(NULL));
	//Segun la dificultad generara unas semillas u otras
	// Si la dificultad es alta
	if (p_dificultad)
	{
		//Semillas que tendra la dificultad alta
		numero_semillas = p_semillas_alta;
		//Inicializamos el tablero
		for (int i = 0; i < p_num_filas; i++)
		{
			for (int j = 0; j < p_num_columnas; j++)
			{
				tablero[i * p_num_columnas + j] = 0;
			}
		}
		//Si hubiera mas semillas que casillas se llena hasta el maximo
		if (numero_semillas > (p_num_columnas*p_num_filas)) {
			numero_semillas = p_num_columnas * p_num_filas;
		}
		//Vector con los indices generados aleatoriamente donde iran los valores
		posicion_semillas = generaIndexSemilla(p_num_columnas, p_num_filas, numero_semillas);
		for (int i = 0; i < numero_semillas; i++)
		{
			//Se coloca en la posicion de la semilla el multiplo aleatorio de 2
			tablero[posicion_semillas[i]] = generarNumeros(p_dificultad);
		}
	}
	//Modo de dificultad baja
	else {
		for (int i = 0; i < p_num_filas; i++)
		{
			for (int j = 0; j < p_num_columnas; j++)
			{
				tablero[i * p_num_columnas + j] = 0;
			}
		}
		//Si hubiera mas semillas que casillas se llena hasta el maximo
		if (numero_semillas > (p_num_columnas*p_num_filas)) {
			numero_semillas = p_num_columnas * p_num_filas;
		}
		//Vector con los indices generados aleatoriamente donde iran los valores
		posicion_semillas = generaIndexSemilla(p_num_columnas, p_num_filas, numero_semillas);
		for (int i = 0; i < numero_semillas; i++)
		{
			//Se coloca en la posicion de la semilla el multiplo aleatorio de 2
			tablero[posicion_semillas[i]] = generarNumeros(p_dificultad);
		}
	}
	//Finalmente devolvemos el tablero
	return tablero;
}
/**
* Funcion que genera multiplos de dos segun la dificultad
*@param dificultad booleano que indica la dificultad del juego
*/
int generarNumeros(bool dificultad) {
	int valoresBajos []= { 2,4,8 };
	int valoresAltos[] = { 2,4};
	//Siempre genera multiplos entre 2, 4 y 8
	int num = valoresBajos[rand() % 3];
	//Si la dificultad es alta los genera entre 2,4
	if (dificultad)
	{
		num = valoresAltos[rand() % 2];
	}
	//Devuelve el multiplo generado
	return num;
}
/**
* Genera un vector con los indices de las semillas para las casillas de la matriz
* @param p_num_columnas columnas de la matriz
* @param p_num_filas filas de la matriz
* @param p_tam tamano del vector a generar
*/
int* generaIndexSemilla(int p_num_columnas, int p_num_filas, int p_tam) {
	//Vector con los indices de las semillas del tamano dado
	int* indexes = (int *)malloc(p_tam * sizeof(int));
	//Primer indice
	int index = rand() % p_num_columnas*p_num_filas;
	//Contador que usaremos en el bucle
	int cont = 0;
	while (cont != p_tam)
	{
		//Si el valor generado no esta se introduce y se busca el siguiente
		if (!contiene(index, indexes, p_tam))
		{
			indexes[cont] = index;
			cont++;
		}
		//Nuevo indice a probar
		index = rand() % (p_num_columnas*p_num_filas);
	}
	//Devuelve los indices de las semillas generadas para la matriz
	return indexes;
}
/**
* Rellena la matriz con multiplos de dos en funcion de la dificultad, semillas y huecos libres
* @param p_tablero tablero que se rellenara con las semillas
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
* @param p_dificultad indica el modo de dificultad elegido
* @param p_semillas_alta numero de semillas que se crearan en nivel de juego alto
* @param p_semillas_baja numero de semillas que se crearan en nivel de juego bajo
*/
int* rellenarTablero(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_alta, int p_semillas_baja, bool* p_lleno) {
	//Huecos disponibles en la matriz
	int huecos = comprobarCasillasVacias(p_tablero, p_num_columnas, p_num_filas);
	//Cantidad de semillas a meter
	int semillas = p_semillas_baja;
	//Indice que se probara
	int index = rand() % (p_num_columnas*p_num_filas);
	//Si es dificil la cantidad cambia
	if (p_dificultad) {
		semillas = p_semillas_alta;
	}
	//Si hay mas huecos que semillas se introducen todas las semillas
	if (semillas < huecos) {
		*p_lleno = false;
		//Mientras tengamos semillas se van buscando posiciones aleatoriamente
		while (semillas > 0) {
			index = rand() % (p_num_columnas*p_num_filas);
			//Se comprueba que este a 0 el hueco
			if (p_tablero[index] == 0) {
				//Se introduce y se reducen las semillas
				p_tablero[index] = generarNumeros(p_dificultad);
				semillas--;
			}
		}
	}
	//Si hay mas semillas que huecos se adaptan
	else {
		*p_lleno = true;
		//Si hay huecos se rellenan si no se acaba
		if (huecos != 0) {
			semillas = huecos;
			//Mientras las nuevas semillas sean mayores que 0 se rellena
			while (semillas > 0) {
				index = rand() % (p_num_columnas*p_num_filas);
				if (p_tablero[index] == 0) {
					p_tablero[index] = generarNumeros(p_dificultad);
					semillas--;
				}
			}
		}
	}
	//Devolvemos el tablero relleno
	return p_tablero;
}

/**
* Comprueba si un valor se encuentra en el vector
* @param p_valor valor que se quiere encontrar
* @param semillas vector donde se buscará
* @param p_tam tamano del vector que hay para buscar
* @return Booleano que indica si esta contenido
*/
bool contiene(int p_valor, int* semillas, int p_tam) {
	//Se incializa el valor por defecto a false
	bool estaContenido = false;
	//Recorremos el vector para comprobar si se encuentra
	for (int i = 0; i < p_tam; i++)
	{
		//Si lo encuentra se para la ejecucion y devuelve true
		if (semillas[i] == p_valor)
		{
			estaContenido = true;
			break;
		}
	}
	//Devuelve el booleano que indica si se contiene
	return estaContenido;
}
/**
* Modalidad de juego en la que funciona automaticamente de forma aleatoria
* @param p_tablero matriz en la que se jugara
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
* @param p_dificultad booleano que indica la dificultad del juego
* @param p_semillas_baja cantidad de semillas que se generan en dificultad baja
* @param p_semillas_alta cantidad de semillas que se generan en dificultad alta
*/
void modoAutomatico(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_baja, int p_semillas_alta)
{
	//Variable para el tamaño
	int size = p_num_columnas * p_num_filas;
	//Semilla que se usara para aleatorizar los numeros aleatorios
	srand((unsigned int)time(NULL));
	int record = cargarRecord();
	int vidas = 5;
	//Cuenta los movimientos que se han hecho
	int cont = 0;
	int puntuacion = 0;
	char tecla;
	int puntuacion_anterior = 0;
	int* puntuacion_dev = 0;
	bool lleno = false;
	bool bloqueoEjeX = false;
	bool bloqueoEjeY = false;
	//Movimiento que se realizara
	int movimiento;
	//Definimos el tablero que habrá en la grafica
	int* tablero_dev;
	//Declaramos la dimension de CUDA
	dim3 dimBlock(p_num_columnas*p_num_filas);
	//Reservamos memoria global en la grafica
	hipMalloc((void**)&tablero_dev, size * sizeof(int));
	hipMalloc((void**)&puntuacion_dev, sizeof(int));
	do {
		//Guardamos la puntuacion anterior para calculos
		puntuacion_anterior = puntuacion;
		//Tiempo de espera para poder apreciar los movimientos
		Sleep(400);
		//Movimiento aleatorio que se crea
		movimiento = rand() % 4;

		//Se limpia la pantalla y se muestra la matriz
		system("cls");
		printf("    ---.::: 16384 - THE GAME :::.---\n\n");
		printf("*************************************\n");
		imprimirMatrizVector(p_tablero, p_num_columnas, p_num_filas);
		printf("*************************************\n");
		printf("Movimiento numero: %d\n", cont);
		printf("Puntuacion: %d\t", puntuacion);
		printf("Record: %d\n", record);
		//Si las vidas llegan a 0 se acaba el juego
		if (vidas < 0) {
			system("cls");
			printf("\n\t-Te has quedado sin vidas-\n");
			printf("\n\t-Has perdido la partida-\n");
			getch();
			//Liberamos la memoria que se emplea en cuda
			hipFree(tablero_dev);
			hipFree(puntuacion_dev);
			if (record < puntuacion) {
				system("cls");
				printf("\n______________________________________________________________________");
				printf("\n -:: Enhorabuena has superado el record guardaremos esta puntuacion ::-");
				printf("\n\n\t\t\t NUEVO RECORD: %d", puntuacion);
				printf("\n\t\t\t ANTERIOR RECORD: %d", record);
				printf("\n______________________________________________________________________\n");
				guardarRecord(puntuacion);
			}
			exit(0);
		}

		//Mostramos las vidas con colores y en funcion de las que quedan
		printf("Vidas: ");
		Color(0, 12);
		for (int i = 0; i < vidas; i++) {
			printf("<3 ");
		}
		Color(0, 15);

		//Segun el valor realiza un movimiento u otro
		switch (movimiento)
		{
			//Movimiento hacia arriba - W
		case 0: printf("\n Direccion: Arriba\n");
			//Se pasa a memoria de la grafica el tablero
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			//Se realiza el calculo en el kernel
			moverArriba << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			//Se devuelve el resultado
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia la izquierda - A
		case 1: printf("\n Direccion: Izquierda\n");
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverIzquierda << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia abajo- S
		case 2: printf("\n Direccion: Abajo\n");
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverAbajo << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia la derecha- D
		case 3: printf("\n Direccion: Derecha\n");
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverDerecha << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
		default:
			break;
		}
		p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
		cont++;

		//---------------------------------COMPROBACION DE BLOQUEO DEL JUEGO----------------------------------------
		//Solo se comprueba cuando esta lleno el tablero para ahorrar costes
		if (lleno)
		{
			//Para comprobar si no se pueden hacer movimientos se comprueban los cambios de puntuaciones
			//Si esta no varia al tener el tablero lleno significa que en ese eje no quedan mas movimientos
			if (puntuacion_anterior == puntuacion)
			{
				//Detectamos el eje en el que no se pueden hacer mas movimientos
				if (movimiento == 0 || movimiento == 2)
				{
					//Se bloquea el eje Y 
					bloqueoEjeY = true;
				}
				else if (movimiento == 1 || movimiento == 3)
				{
					//Se bloquea el eje X
					bloqueoEjeX = true;
				}
			}
			//Si las puntuaciones difieren aunque este lleno el tablero significa que aun hay movimientos
			else {
				//Se desbloquean los dos ejes
				bloqueoEjeX = false;
				bloqueoEjeY = false;
			}

		}
		//Si no esta lleno significa que hay huecos para hacer movimientos
		else {
			//Se desbloquean los dos ejes
			bloqueoEjeX = false;
			bloqueoEjeY = false;
		}
		//Si se bloquean los dos ejes simultaneamente el juego se bloquea
		if (bloqueoEjeX && bloqueoEjeY)
		{
			//Desbloqueamos ejes por si se quiere continuar jugando 
			bloqueoEjeX = false;
			bloqueoEjeY = false;
			//Se muestran los mensajes
			printf("\nHAS PERDIDO\n");
			printf("¿Quieres volver a jugar? (Pulsa y para jugar)\n");
			//Obtenemos la respuesta
			scanf(" %c", &tecla);
			//Si responde afirmativamente se quita un vida y se reinicia el tablero
			if (tecla == 'y' || tecla == 'Y')
			{
				vidas--;
				p_tablero = inicializarMatriz(p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja);
			}
			//Si no quiere jugar se le pregunta si desea guardar la partida para continuarla despues
			else {
				printf("¿Desea guardar la partida? (Pulsa y para guardar)\n");
				//Obtenemos la respuesta
				scanf(" %c", &tecla);
				//Si responde afirmativamente se guardan los datos y se sale
				if (tecla == 'y' || tecla == 'Y')
				{
					guardarPartida(p_tablero, vidas, puntuacion, p_num_columnas, p_num_filas);
					printf("¡Se han guardado los datos correctamente!\n");
				}
				printf("Saliendo...");
				//Liberamos la memoria que se emplea en cuda
				hipFree(tablero_dev);
				hipFree(puntuacion_dev);
				//Codigo de exito 0
				if (record < puntuacion) {
					system("cls");
					printf("\n______________________________________________________________________");
					printf("\n -:: Enhorabuena has superado el record guardaremos esta puntuacion ::-");
					printf("\n\n\t\t\t NUEVO RECORD: %d", puntuacion);
					printf("\n\t\t\t ANTERIOR RECORD: %d", record);
					printf("\n______________________________________________________________________\n");
					guardarRecord(puntuacion);
				}
				exit(0);
			}

		}
	} while (true);
}
/**
* Modalidad de juego en la que funciona manualmente 
* @param p_tablero matriz en la que se jugara
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
* @param p_dificultad booleano que indica la dificultad del juego
* @param p_semillas_baja cantidad de semillas que se generan en dificultad baja
* @param p_semillas_alta cantidad de semillas que se generan en dificultad alta
*/
void modoManual(int* p_tablero, int p_num_columnas, int p_num_filas, bool p_dificultad, int p_semillas_baja, int p_semillas_alta)
{

	//Declaracion e inicializacion de variables
	int record = cargarRecord();
	int vidas = 5;
	int size = p_num_columnas * p_num_filas;
	int puntuacion = 0;
	char tecla;
	int puntuacion_anterior = 0;
	//Booleanos empleados en el control del bloqueo del juego
	bool bloqueoEjeX = false;
	bool bloqueoEjeY = false;
	bool lleno = false;
	//-------------------------------CARGA DE PARTIDA----------------------------------------
	//Limpiamos la pantalla
	system("cls");
	//Preguntamos si desea cargar una partida
	printf("Desea cargar partida?(Pulse y para cargar partida)\n");
	tecla = getch();
	//Si responde afirmativamente se reemplazan los valores por los almacenados
	if (tecla == 'y') {
		p_tablero = cargarPartida(&vidas, &puntuacion, &p_num_columnas, &p_num_filas);
		//Se actualiza el tamaño a reservar
		size = p_num_columnas * p_num_filas;
	}
	//--------------------------Declaracion y reserva para CUDA------------------------------

	//Definimos el tablero que habrá en la grafica
	int* puntuacion_dev = 0;
	int* tablero_dev;
	//Declaramos la dimension de CUDA
	dim3 dimBlock(p_num_columnas*p_num_filas);
	//Reservamos memoria global en la grafica
	hipMalloc((void**)&tablero_dev, size * sizeof(int));
	hipMalloc((void**)&puntuacion_dev, sizeof(int));
	//--------------------------- BUCLE DEL JUEGO---------------------------------------
	do {
		//Guardamos la puntuacion anterior para calculos
		puntuacion_anterior = puntuacion;

		//Mostramos los datos por pantalla
		system("cls");
		printf("    ---.::: 16384 - THE GAME :::.---\n\n");
		printf("==============================================\n");
		imprimirMatrizVector(p_tablero, p_num_columnas, p_num_filas);
		printf("==============================================\n");
		printf("Puntuacion: %d\t", puntuacion);
		printf("Record: %d\n", record);

		//Si las vidas son menores a 0 se acaba el juego
		if (vidas < 0) {
			printf("\n\t-Te has quedado sin vidas-\n");
			printf("\n\t-Has perdido la partida-\n");
			getch();
			//Liberamos la memoria que se emplea en cuda
			hipFree(tablero_dev);
			hipFree(puntuacion_dev);
			if (record < puntuacion) {
				printf("\n -Enhorabuena has superado el record guardaremos esta puntuacion-");
				guardarRecord(puntuacion);
			}
			exit(0);
		}
		//Mostramos las vidas con colores y segun cantidad
		printf("Vidas: ");
		Color(0, 12);
		for (int i = 0; i < vidas; i++) {
			printf("<3 ");
		}
		Color(0, 15);

		//Solicitamos un movimiento
		tecla = getch();
		//En el caso de las flechas para evitar errores se solicita 2 veces
		if (tecla == -32) {
			tecla = getch();
		}
		switch (tecla)
		{
			//Movimiento hacia arriba - W
		case 'w':
			//Se pasa a memoria de la grafica el tablero
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			//Se realiza el calculo en el kernel
			moverArriba << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			//Se devuelve el resultado
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
		case 'W':
			//Se pasa a memoria de la grafica el tablero
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			//Se realiza el calculo en el kernel
			moverArriba << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			//Se devuelve el resultado
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			//Despues de realizar el movimiento se rellena con valores
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Se corresponde con la flecha del teclado
		case 72:
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverArriba << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia la izquierda - A
		case 'a':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverIzquierda << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
		case 'A':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverIzquierda << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Se corresponde con la flecha del teclado
		case 75:
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverIzquierda << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia abajo- S
		case 's':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverAbajo << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
		case 'S':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverAbajo << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Se corresponde con la flecha del teclado
		case 80:
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverAbajo << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Movimiento hacia la derecha- D
		case 'd':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverDerecha << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
		case 'D':
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverDerecha << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Se corresponde con la flecha del teclado
		case 77:
			hipMemcpy(tablero_dev, p_tablero, size * sizeof(int), hipMemcpyHostToDevice);
			moverDerecha << <1, dimBlock >> > (tablero_dev, p_num_columnas, p_num_filas, puntuacion_dev);
			hipMemcpy(p_tablero, tablero_dev, size * sizeof(int), hipMemcpyDeviceToHost);
			//Obtenemos la puntuacion despues del movimiento
			hipMemcpy(&puntuacion, puntuacion_dev, sizeof(int), hipMemcpyDeviceToHost);
			p_tablero = rellenarTablero(p_tablero, p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja, &lleno);
			break;
			//Guardamos los datos de la partida
		case 'g': printf("Guardando partida...\n");
			guardarPartida(p_tablero, vidas, puntuacion, p_num_columnas, p_num_filas);
			break;
		case 'G': printf("Guardando partida...\n");
			guardarPartida(p_tablero, vidas, puntuacion, p_num_columnas, p_num_filas);
			break;
			//Se sale del juego
		case 'e': printf("Saliendo ...\n");
			//Liberamos la memoria que se emplea
			hipFree(tablero_dev);
			hipFree(puntuacion_dev);
			if (record < puntuacion) {
				system("cls");
				printf("\n______________________________________________________________________");
				printf("\n -:: Enhorabuena has superado el record guardaremos esta puntuacion ::-");
				printf("\n\n\t\t\t NUEVO RECORD: %d", puntuacion);
				printf("\n\t\t\t ANTERIOR RECORD: %d", record);
				printf("\n______________________________________________________________________\n");
				guardarRecord(puntuacion);
			}
			exit(0);
			break;
		case 'E': printf("Saliendo ...\n");
			//Liberamos la memoria que se emplea
			hipFree(tablero_dev);
			hipFree(puntuacion_dev);
			if (record < puntuacion) {
				system("cls");
				printf("\n______________________________________________________________________");
				printf("\n -:: Enhorabuena has superado el record guardaremos esta puntuacion ::-");
				printf("\n\n\t\t\t NUEVO RECORD: %d", puntuacion);
				printf("\n\t\t\t ANTERIOR RECORD: %d", record);
				printf("\n______________________________________________________________________\n");
				guardarRecord(puntuacion);
			}
			exit(0);
			break;
		default:
			break;
		}

		//---------------------------------COMPROBACION DE BLOQUEO DEL JUEGO----------------------------------------
		//Solo se comprueba cuando esta lleno el tablero para ahorrar costes
		if (lleno)
		{
			//Para comprobar si no se pueden hacer movimientos se comprueban los cambios de puntuaciones
			//Si esta no varia al tener el tablero lleno significa que en ese eje no quedan mas movimientos
			if (puntuacion_anterior == puntuacion)
			{
				//Detectamos el eje en el que no se pueden hacer mas movimientos
				if (tecla == 'w' || tecla == 's' || tecla == 'W' || tecla == 'S' || tecla == 72 || tecla == 80)
				{
					//Se bloquea el eje Y 
					bloqueoEjeY = true;
				}
				else if (tecla == 'a' || tecla == 'd' || tecla == 'A' || tecla == 'D' || tecla == 77 || tecla == 75)
				{
					//Se bloquea el eje X
					bloqueoEjeX = true;
				}
			}
			//Si las puntuaciones difieren aunque este lleno el tablero significa que aun hay movimientos
			else {
				//Se desbloquean los dos ejes
				bloqueoEjeX = false;
				bloqueoEjeY = false;
			}

		}
		//Si no esta lleno significa que hay huecos para hacer movimientos
		else {
			//Se desbloquean los dos ejes
			bloqueoEjeX = false;
			bloqueoEjeY = false;
		}
		//Si se bloquean los dos ejes simultaneamente el juego se bloquea
		if (bloqueoEjeX && bloqueoEjeY)
		{
			//Desbloqueamos ejes por si se quiere continuar jugando 
			bloqueoEjeX = false;
			bloqueoEjeY = false;
			//Se muestran los mensajes
			printf("\nHAS PERDIDO\n");
			printf("¿Quieres volver a jugar? (Pulsa y para jugar)\n");
			//Obtenemos la respuesta
			scanf(" %c", &tecla);
			//Si responde afirmativamente se quita un vida y se reinicia el tablero
			if (tecla == 'y' || tecla == 'Y')
			{
				vidas--;
				p_tablero = inicializarMatriz(p_num_columnas, p_num_filas, p_dificultad, p_semillas_alta, p_semillas_baja);
			}
			//Si no quiere jugar se le pregunta si desea guardar la partida para continuarla despues
			else {
				printf("¿Desea guardar la partida? (Pulsa y para guardar)\n");
				//Obtenemos la respuesta
				scanf(" %c", &tecla);
				//Si responde afirmativamente se guardan los datos y se sale
				if (tecla == 'y' || tecla == 'Y')
				{
					guardarPartida(p_tablero, vidas, puntuacion, p_num_columnas, p_num_filas);
					printf("¡Se han guardado los datos correctamente!\n");
				}
				printf("Saliendo...");
				//Liberamos la memoria que se emplea en cuda
				hipFree(tablero_dev);
				hipFree(puntuacion_dev);
				//Codigo de exito 0
				if (record < puntuacion) {
					system("cls");
					printf("\n______________________________________________________________________");
					printf("\n -:: Enhorabuena has superado el record guardaremos esta puntuacion ::-");
					printf("\n\n\t\t\t NUEVO RECORD: %d", puntuacion);
					printf("\n\t\t\t ANTERIOR RECORD: %d", record);
					printf("\n______________________________________________________________________\n");
					guardarRecord(puntuacion);
				}
				exit(0);
			}

		}
	} while (true);
}

//--------------------------------CARGAR Y GUARDAR PARTIDAS--------------------------------------

/**
* Guarda el estado de la partida con su tablero y datos
* @param p_tablero guarda la matriz completa
* @param p_vidas numero de vidas disponibles
* @param p_puntuacion puntuacion que se tenia en el momento del guardado
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
*/
void guardarPartida(int* p_tablero, int p_vidas, int p_puntuacion, int p_num_columnas, int p_num_filas) {
	FILE *archivo;
	//Se abre/crea el archivo para escribir
	archivo = fopen("datos_partida.txt", "w");
	//En caso de que se haya hecho bien se procede al guardado
	if (archivo != NULL) {
		//Guardamos cada variable, y la matriz la escribimos completa
		fprintf(archivo, "%i \n", p_vidas);
		fprintf(archivo, "%i \n", p_puntuacion);
		fprintf(archivo, "%i \n", p_num_columnas);
		fprintf(archivo, "%i \n", p_num_filas);
		for (int i = 0; i < (p_num_columnas*p_num_filas); i++) {
			fprintf(archivo, "%i ", p_tablero[i]);
		}
		//Cerramos el archivo
		fclose(archivo);
		printf("Guardado realizado con exito.\n");
	}
	//Si se ha creado mal se muestra un mensaje
	else {
		printf("No se ha podido abrir/crear el archivo\n");
		exit(-1);
	}
}
/**
* Carga el estado de la partida con su tablero y datos mediante punteros
* @param p_vidas numero de vidas disponibles
* @param p_puntuacion puntuacion que se tenia en el momento del guardado
* @param p_num_columnas Anchura de la matriz
* @param p_num_filas Altura de la matriz
* @return devuelve el tablero
*/
int* cargarPartida(int* p_vidas, int* p_puntuacion, int* p_num_columnas, int* p_num_filas) {
	//Declaramos variables
	int* tablero;
	FILE *archivo;
	//Indicamos el archivo a leer
	archivo = fopen("datos_partida.txt", "r");
	//Leera los datos solo si se ha abierto bien el archivo
	if (archivo != NULL) {
		//Se leen las variables y la matriz a partir de sus variables
		fscanf(archivo, "%i", p_vidas);
		fscanf(archivo, "%i", p_puntuacion);
		fscanf(archivo, "%i", p_num_columnas);
		fscanf(archivo, "%i", p_num_filas);
		//Inicializamos la matriz tablero
		tablero = (int *)malloc((*p_num_columnas)*(*p_num_filas) * sizeof(int));
		//Recomponemos la matriz tablero a partir de los elementos
		for (int i = 0; i < (*p_num_columnas)*(*p_num_filas); i++) {
			fscanf(archivo, "%i", &tablero[i]);
		}
		//Cerramos el archivo
		fclose(archivo);
		printf("Partida cargada con exito.\n");
	}
	else {
		printf("No se ha podido leer el archivo, es posible que no haya ninguna guardada\n");
		exit(-1);
	}
	//Se devuelve el tablero finalmente
	return tablero;
}
/**
* Guarda la puntuacion record
* @param p_record nuevo el record del juego a guardar
*/
void guardarRecord(int p_record) {
	FILE *archivo;
	//Se abre/crea el archivo para escribir
	archivo = fopen("record.txt", "w");
	//En caso de que se haya hecho bien se procede al guardado
	if (archivo != NULL) {
		//Guardamos la variable record
		fprintf(archivo, "%i \n", p_record);
		//Cerramos el archivo
		fclose(archivo);
		printf("\nGuardado realizado con exito.\n");
	}
	//Si se ha creado mal se muestra un mensaje
	else { printf("No se ha podido abrir/crear el archivo\n"); }
}
/**
* Carga la puntuacion record
* @return devuelve el record del juego
*/
int cargarRecord() {
	//Declaramos variables
	int record = 0;
	FILE *archivo;
	//Indicamos el archivo a leer
	archivo = fopen("record.txt", "r");
	//Leera los datos solo si se ha abierto bien el archivo
	if (archivo != NULL) {
		//Se lee la variable
		fscanf(archivo, "%i", &record);
		//Cerramos el archivo
		fclose(archivo);
	}
	else { printf("\nNo hay ningun record guardado por tanto el record actual es 0"); }
	//Se devuelve el record, sera 0 si no hay ninguno
	return record;
}
/**
* Comprueba las casillas vacias que hay y devuelve su numero
* @param p_tablero tablero que se comprobara
* @param p_num_columnas numero de columnas del tablero
* @param p_num_filas numero de filas del tablero
*/
int comprobarCasillasVacias(int * p_tablero, int p_num_columnas, int p_num_filas) {
	int casillas = 0;
	//Se recorre el tablero contando las casillas cuyo valor es 0
	for (int i = 0; i < p_num_filas; i++)
	{
		for (int j = 0; j < p_num_columnas; j++)
		{
			if (p_tablero[i*p_num_columnas + j] == 0)
			{
				//Si se encuentra una cuyo valor es 0 se añade a la cuenta
				casillas++;
			}
			
		}
	}
	//Devolvemos el total de baldosas vacias
	return casillas;
}
/**
* Obtiene las caracteristicas de la grafica e indica si puede correr la matriz
* @param p_num_columnas numero de columnas del tablero
* @param p_num_filas numero de filas del tablero
*/
void obtenerCaracteristicasCUDA(int p_num_columnas, int p_num_filas) {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("Las caracteristicas de su grafica son las siguientes:\n");
	//Version y nombre de la grafica
	printf("Nombre: %s - Capability Version: %d.%d\n", prop.name, prop.major, prop.minor);
	printf("Limites de hilos por bloque: %d\n", prop.maxThreadsPerBlock);
	printf("Limites de hilos por SM: %d\n", prop.maxThreadsPerMultiProcessor);
	printf("Limites de memoria global: %zd\n B", prop.totalGlobalMem);
	printf("Limites de memoria compartida: %zd\n B", prop.sharedMemPerBlock);
	printf("Limites de registros: %d\n B", prop.regsPerBlock);
	printf("Numero de multiprocesadores: %d\n", prop.multiProcessorCount);
	//Caracteristicas de la matriz en memoria global sin teselar ni usar compartida
	printf("Las caracteristicas de la matriz a emplear:\n");
	printf("Cantidad de hilos a emplear: %d\n", p_num_columnas*p_num_filas);
	//Se empleara la matriz de enteros y un entero mas para la suma de puntuacion
	printf("Cantidad de memoria que se emplea: %zd\n", p_num_columnas*p_num_filas * sizeof(int) + sizeof(int));
	//Se comprueba que no se exceda el maximo de hilos por bloque para ver si puede con la matriz
	if (prop.maxThreadsPerBlock < p_num_columnas*p_num_filas) {
		printf("Error no hay suficientes hilos para ejecutar la matriz");
		exit(-1);
	}
	//Se comprueba que no se exceda el maximo de memoria global para ver si puede con la matriz
	if (prop.totalGlobalMem < p_num_columnas*p_num_filas * sizeof(int)) {
		printf("Error no hay suficiente memoria global para ejecutar la matriz");
		exit(-1);
	}
	//Comparacion en porcentajes
	printf("Porcentaje de aprovechamiento de hilos: %.2f %% \n", ((double)(p_num_columnas*p_num_filas) / prop.maxThreadsPerBlock) * 100);
}
/**
* Permite cambiar el color del texto y fondo de la consola
* @param p_fondo entero que permitira seleccionar un color para el fondo
* @param p_fuente entero que permitira seleccionar un color para el texto
*/
void Color(int p_fondo, int p_fuente) {

	HANDLE Consola = GetStdHandle(STD_OUTPUT_HANDLE);
	//Cálculo para convertir los colores al valor necesario
	int color_nuevo = p_fuente + (p_fondo * 16);
	//Aplicamos el color a la consola
	SetConsoleTextAttribute(Consola, color_nuevo);
}